#include "hip/hip_runtime.h"
#include "PCFG_GPU.h"
#include <hip/hip_runtime.h>
#include <string>
#include <thread>
#include <mutex>
#include <cstring>

using namespace std;
//nvcc main.cpp train.cpp guessing_GPU.cu md5.cpp -o GPU.exe

// 用于线程间同步的互斥锁 - 使用读写锁优化
static std::mutex gpu_mutex;

// 预计算的PT工作量缓存，避免重复计算
static std::unordered_map<int, int> pt_workload_cache;
static std::mutex cache_mutex;

// 单段口令生成 kernel - 采用guessing.cu的简洁设计
__global__ void kernel_single(const char *d_values, char *d_out, int num, int maxlen) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num) {
        const char *src = d_values + idx * maxlen;
        char *dst = d_out + idx * maxlen;
        int i = 0;
        for (; i < maxlen - 1 && src[i] != '\0'; ++i) dst[i] = src[i];
        dst[i] = '\0';
    }
}

// 多段口令生成 kernel - 采用guessing.cu的简洁设计
__global__ void kernel_multi(const char *prefix, int prefix_len, const char *d_values, char *d_out, int num, int maxlen) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num) {
        char *dst = d_out + idx * maxlen;
        int i = 0;
        for (; i < prefix_len && i < maxlen - 1; ++i) dst[i] = prefix[i];
        const char *src = d_values + idx * maxlen;
        int j = 0;
        for (; i < maxlen - 1 && src[j] != '\0'; ++i, ++j) dst[i] = src[j];
        dst[i] = '\0';
    }
}

// GPU并行单段生成 - 完全采用guessing.cu的实现
void gpu_generate_single(const vector<string> &values, vector<string> &guesses) {
    int num = values.size();
    if (num == 0) return;
    int maxlen = 0;
    for (auto &s : values) maxlen = max(maxlen, (int)s.size());
    maxlen += 1;
    vector<char> h_values(num * maxlen, 0);
    for (int i = 0; i < num; ++i)
        strncpy(&h_values[i * maxlen], values[i].c_str(), maxlen);

    char *d_values, *d_out;
    hipMalloc(&d_values, num * maxlen);
    hipMalloc(&d_out, num * maxlen);
    hipMemcpy(d_values, h_values.data(), num * maxlen, hipMemcpyHostToDevice);

    int block = 256, grid = (num + block - 1) / block;
    kernel_single<<<grid, block>>>(d_values, d_out, num, maxlen);
    hipDeviceSynchronize();

    vector<char> h_out(num * maxlen, 0);
    hipMemcpy(h_out.data(), d_out, num * maxlen, hipMemcpyDeviceToHost);

    guesses.clear();
    for (int i = 0; i < num; ++i)
        guesses.emplace_back(&h_out[i * maxlen]);

    hipFree(d_values);
    hipFree(d_out);
}

// GPU并行多段生成 - 完全采用guessing.cu的实现
void gpu_generate_multi(const string &prefix, const vector<string> &values, vector<string> &guesses) {
    int num = values.size();
    if (num == 0) return;
    int maxlen = prefix.size();
    for (auto &s : values) maxlen = max(maxlen, (int)(prefix.size() + s.size()));
    maxlen += 1;
    vector<char> h_values(num * maxlen, 0);
    for (int i = 0; i < num; ++i)
        strncpy(&h_values[i * maxlen], values[i].c_str(), maxlen);

    char *d_values, *d_out, *d_prefix;
    hipMalloc(&d_values, num * maxlen);
    hipMalloc(&d_out, num * maxlen);
    hipMalloc(&d_prefix, maxlen);
    hipMemcpy(d_values, h_values.data(), num * maxlen, hipMemcpyHostToDevice);
    hipMemcpy(d_prefix, prefix.c_str(), prefix.size(), hipMemcpyHostToDevice);

    int block = 256, grid = (num + block - 1) / block;
    kernel_multi<<<grid, block>>>(d_prefix, prefix.size(), d_values, d_out, num, maxlen);
    hipDeviceSynchronize();

    vector<char> h_out(num * maxlen, 0);
    hipMemcpy(h_out.data(), d_out, num * maxlen, hipMemcpyDeviceToHost);

    guesses.clear();
    for (int i = 0; i < num; ++i)
        guesses.emplace_back(&h_out[i * maxlen]);

    hipFree(d_values);
    hipFree(d_out);
    hipFree(d_prefix);
}

// 添加缺失的函数实现

void PriorityQueue::CalProb(PT &pt)
{
    // 计算一个PT本身的概率。后续所有具体segment value的概率，直接累乘在这个初始概率值上
    pt.prob = pt.preterm_prob;

    // index: 标注当前segment在PT中的位置
    int index = 0;

    for (int idx : pt.curr_indices)
    {
        if (pt.content[index].type == 1)
        {
            pt.prob *= m.letters[m.FindLetter(pt.content[index])].ordered_freqs[idx];
            pt.prob /= m.letters[m.FindLetter(pt.content[index])].total_freq;
        }
        if (pt.content[index].type == 2)
        {
            pt.prob *= m.digits[m.FindDigit(pt.content[index])].ordered_freqs[idx];
            pt.prob /= m.digits[m.FindDigit(pt.content[index])].total_freq;
        }
        if (pt.content[index].type == 3)
        {
            pt.prob *= m.symbols[m.FindSymbol(pt.content[index])].ordered_freqs[idx];
            pt.prob /= m.symbols[m.FindSymbol(pt.content[index])].total_freq;
        }
        index += 1;
    }
}

void PriorityQueue::init()
{
    // 用所有可能的PT，按概率降序填满整个优先队列
    for (PT pt : m.ordered_pts)
    {
        for (segment seg : pt.content)
        {
            if (seg.type == 1)
            {
                pt.max_indices.emplace_back(m.letters[m.FindLetter(seg)].ordered_values.size());
            }
            if (seg.type == 2)
            {
                pt.max_indices.emplace_back(m.digits[m.FindDigit(seg)].ordered_values.size());
            }
            if (seg.type == 3)
            {
                pt.max_indices.emplace_back(m.symbols[m.FindSymbol(seg)].ordered_values.size());
            }
        }
        pt.preterm_prob = float(m.preterm_freq[m.FindPT(pt)]) / m.total_preterm;

        // 计算当前pt的概率
        CalProb(pt);
        // 将PT放入优先队列
        priority.emplace_back(pt);
    }
}

vector<PT> PT::NewPTs()
{
    // 存储生成的新PT
    vector<PT> res;

    // 假如这个PT只有一个segment
    if (content.size() == 1)
    {
        return res;
    }
    else
    {
        // 最初的pivot值
        int init_pivot = pivot;

        // 开始遍历所有位置值大于等于init_pivot值的segment
        for (int i = pivot; i < curr_indices.size() - 1; i += 1)
        {
            curr_indices[i] += 1;

            if (curr_indices[i] < max_indices[i])
            {
                PT new_pt = *this;
                new_pt.pivot = i;
                res.emplace_back(new_pt);
                curr_indices[i] -= 1;
                break;
            }
            else
            {
                curr_indices[i] = 0;
                if (i == curr_indices.size() - 2)
                {
                    pivot = init_pivot;
                    return res;
                }
            }

            curr_indices[i] -= 1;
        }
        pivot = init_pivot;
        return res;
    }

    return res;
}

// 优化：缓存工作量计算，避免重复查找
int getLastSegmentWorkload(PriorityQueue* queue, PT& pt) {
    // 简单的缓存策略：使用PT内容的哈希作为缓存键
    int cache_key = pt.content.size() * 1000 + pt.content.back().type * 100 + pt.content.back().length;
    
    {
        std::lock_guard<std::mutex> lock(cache_mutex);
        auto it = pt_workload_cache.find(cache_key);
        if (it != pt_workload_cache.end()) {
            return it->second;
        }
    }
    
    // 计算工作量
    int lastSegmentIndex = pt.content.size() - 1;
    int valueCount = 0;
    
    if (lastSegmentIndex >= 0) {
        segment* lastSeg;
        if (pt.content[lastSegmentIndex].type == 1) {
            lastSeg = &queue->m.letters[queue->m.FindLetter(pt.content[lastSegmentIndex])];
        } else if (pt.content[lastSegmentIndex].type == 2) {
            lastSeg = &queue->m.digits[queue->m.FindDigit(pt.content[lastSegmentIndex])];
        } else {
            lastSeg = &queue->m.symbols[queue->m.FindSymbol(pt.content[lastSegmentIndex])];
        }
        valueCount = lastSeg->ordered_values.size();
    }
    
    // 缓存结果
    {
        std::lock_guard<std::mutex> lock(cache_mutex);
        pt_workload_cache[cache_key] = valueCount;
    }
    
    return valueCount;
}

// 优化：降低GPU阈值，并加入简单的自适应机制
bool PriorityQueueExtension::isGPUSuitable(PriorityQueue* queue, PT& pt) {
    int valueCount = getLastSegmentWorkload(queue, pt);
    
    // 动态阈值：根据GPU利用率调整
    static int gpu_threshold = 500;  // 初始阈值降低
    static int call_count = 0;
    static int gpu_calls = 0;
    
    call_count++;
    if (valueCount > gpu_threshold) {
        gpu_calls++;
    }
    
    // 每1000次调用调整一次阈值
    if (call_count % 1000 == 0) {
        double gpu_ratio = (double)gpu_calls / call_count;
        if (gpu_ratio < 0.1) {  // GPU使用率太低，降低阈值
            gpu_threshold = max(200, gpu_threshold - 100);
        } else if (gpu_ratio > 0.5) {  // GPU使用率太高，提高阈值
            gpu_threshold = min(2000, gpu_threshold + 100);
        }
    }
    
    return valueCount > gpu_threshold;
}

// 优化：减少锁争用，使用批量插入
void PriorityQueueExtension::GenerateCPU(PriorityQueue* queue, PT pt) {
    // 计算PT的概率
    queue->CalProb(pt);

    // 对于只有一个segment的PT
    if (pt.content.size() == 1) {
        segment *a;
        if (pt.content[0].type == 1) {
            a = &queue->m.letters[queue->m.FindLetter(pt.content[0])];
        } else if (pt.content[0].type == 2) {
            a = &queue->m.digits[queue->m.FindDigit(pt.content[0])];
        } else {
            a = &queue->m.symbols[queue->m.FindSymbol(pt.content[0])];
        }
        
        // 关键：完全按照guessing.cu的思路，直接使用所有ordered_values
        const vector<string>& values = a->ordered_values;  // 使用引用避免拷贝
        
        // 优化：批量插入，减少锁争用
        {
            std::lock_guard<std::mutex> lock(gpu_mutex);
            queue->guesses.reserve(queue->guesses.size() + values.size());  // 预留空间
            queue->guesses.insert(queue->guesses.end(), values.begin(), values.end());
            queue->total_guesses += values.size();
        }
    } else {
        string guess;
        int seg_idx = 0;
        // 构建前缀
        for (size_t k = 0; k < pt.curr_indices.size() && seg_idx < (int)pt.content.size() - 1; k++) {
            int idx = pt.curr_indices[k];
            if (pt.content[seg_idx].type == 1) {
                guess += queue->m.letters[queue->m.FindLetter(pt.content[seg_idx])].ordered_values[idx];
            } else if (pt.content[seg_idx].type == 2) {
                guess += queue->m.digits[queue->m.FindDigit(pt.content[seg_idx])].ordered_values[idx];
            } else if (pt.content[seg_idx].type == 3) {
                guess += queue->m.symbols[queue->m.FindSymbol(pt.content[seg_idx])].ordered_values[idx];
            }
            seg_idx += 1;
        }

        // 获取最后一个segment
        segment *a;
        if (pt.content[pt.content.size() - 1].type == 1) {
            a = &queue->m.letters[queue->m.FindLetter(pt.content[pt.content.size() - 1])];
        } else if (pt.content[pt.content.size() - 1].type == 2) {
            a = &queue->m.digits[queue->m.FindDigit(pt.content[pt.content.size() - 1])];
        } else {
            a = &queue->m.symbols[queue->m.FindSymbol(pt.content[pt.content.size() - 1])];
        }
        
        // 关键：完全按照guessing.cu的思路，直接使用所有ordered_values
        const vector<string>& values = a->ordered_values;  // 使用引用避免拷贝
        vector<string> localGuesses;
        localGuesses.reserve(values.size());  // 预留空间
        
        // 生成所有组合
        for (const string& value : values) {
            localGuesses.emplace_back(guess + value);
        }
        
        // 优化：批量插入
        {
            std::lock_guard<std::mutex> lock(gpu_mutex);
            queue->guesses.reserve(queue->guesses.size() + localGuesses.size());  // 预留空间
            queue->guesses.insert(queue->guesses.end(), localGuesses.begin(), localGuesses.end());
            queue->total_guesses += localGuesses.size();
        }
    }
}

// 优化：GPU处理也使用类似优化
void PriorityQueueExtension::GenerateGPU(PriorityQueue* queue, PT pt) {
    // 计算PT的概率
    queue->CalProb(pt);
    
    // 对于只有一个segment的PT
    if (pt.content.size() == 1) {
        segment *a;
        if (pt.content[0].type == 1) {
            a = &queue->m.letters[queue->m.FindLetter(pt.content[0])];
        } else if (pt.content[0].type == 2) {
            a = &queue->m.digits[queue->m.FindDigit(pt.content[0])];
        } else {
            a = &queue->m.symbols[queue->m.FindSymbol(pt.content[0])];
        }
        
        // 关键：完全按照guessing.cu的思路，直接使用所有ordered_values
        const vector<string>& values = a->ordered_values;  // 使用引用避免拷贝
        vector<string> local_guesses;
        gpu_generate_single(values, local_guesses);
        
        // 优化：批量插入
        {
            std::lock_guard<std::mutex> lock(gpu_mutex);
            queue->guesses.reserve(queue->guesses.size() + local_guesses.size());  // 预留空间
            queue->guesses.insert(queue->guesses.end(), local_guesses.begin(), local_guesses.end());
            queue->total_guesses += local_guesses.size();
        }
    } else {
        string guess;
        int seg_idx = 0;
        // 构建前缀
        for (size_t k = 0; k < pt.curr_indices.size() && seg_idx < (int)pt.content.size() - 1; k++) {
            int idx = pt.curr_indices[k];
            if (pt.content[seg_idx].type == 1) {
                guess += queue->m.letters[queue->m.FindLetter(pt.content[seg_idx])].ordered_values[idx];
            } else if (pt.content[seg_idx].type == 2) {
                guess += queue->m.digits[queue->m.FindDigit(pt.content[seg_idx])].ordered_values[idx];
            } else if (pt.content[seg_idx].type == 3) {
                guess += queue->m.symbols[queue->m.FindSymbol(pt.content[seg_idx])].ordered_values[idx];
            }
            seg_idx += 1;
        }

        // 获取最后一个segment
        segment *a;
        if (pt.content[pt.content.size() - 1].type == 1) {
            a = &queue->m.letters[queue->m.FindLetter(pt.content[pt.content.size() - 1])];
        } else if (pt.content[pt.content.size() - 1].type == 2) {
            a = &queue->m.digits[queue->m.FindDigit(pt.content[pt.content.size() - 1])];
        } else {
            a = &queue->m.symbols[queue->m.FindSymbol(pt.content[pt.content.size() - 1])];
        }
        
        // 关键：完全按照guessing.cu的思路，直接使用所有ordered_values
        const vector<string>& values = a->ordered_values;  // 使用引用避免拷贝
        vector<string> local_guesses;
        gpu_generate_multi(guess, values, local_guesses);
        
        // 优化：批量插入
        {
            std::lock_guard<std::mutex> lock(gpu_mutex);
            queue->guesses.reserve(queue->guesses.size() + local_guesses.size());  // 预留空间
            queue->guesses.insert(queue->guesses.end(), local_guesses.begin(), local_guesses.end());
            queue->total_guesses += local_guesses.size();
        }
    }
}

// 混合方式生成口令（GPU+CPU协同工作）
void PriorityQueueExtension::GenerateHybrid(PriorityQueue* queue, PT pt) {
    // 根据工作量决定使用GPU还是CPU
    if (isGPUSuitable(queue, pt)) {
        // 使用GPU处理
        GenerateGPU(queue, pt);
    } else {
        // 使用CPU处理
        GenerateCPU(queue, pt);
    }
}

// 进阶功能：批量处理优化
void PriorityQueueExtension::GenerateBatch(PriorityQueue* queue, vector<PT>& pts, int batchSize) {
    // 预分类和预排序优化
    vector<PT> gpuPTs, cpuPTs;
    gpuPTs.reserve(pts.size() / 2);  // 预留空间
    cpuPTs.reserve(pts.size() / 2);
    
    for (PT& pt : pts) {  // 使用引用避免拷贝
        if (isGPUSuitable(queue, pt)) {
            gpuPTs.push_back(std::move(pt));  // 使用移动语义
        } else {
            cpuPTs.push_back(std::move(pt));
        }
    }
    
    // 优化：并行处理GPU和CPU任务
    std::thread gpu_thread([&]() {
        for (PT& pt : gpuPTs) {
            GenerateGPU(queue, pt);
        }
    });
    
    // CPU任务在主线程处理
    for (PT& pt : cpuPTs) {
        GenerateCPU(queue, pt);
    }
    
    gpu_thread.join();
}

// 原始的Generate函数，保留向后兼容性
void PriorityQueue::Generate(PT pt) {
    // 使用混合模式
    PriorityQueueExtension::GenerateHybrid(this, pt);
}

// 扩展原有的PopNext函数 - 使用简化版本
void PriorityQueue::PopNext() {
    // 恢复原始逻辑：完全处理单个PT
    PriorityQueueExtension::GenerateHybrid(this, priority.front());

    // 然后需要根据即将出队的PT，生成一系列新的PT
    vector<PT> new_pts = priority.front().NewPTs();
    for (size_t i = 0; i < new_pts.size(); i++) {
        PT pt = new_pts[i];
        // 计算概率
        CalProb(pt);
        // 接下来的这个循环，作用是根据概率，将新的PT插入到优先队列中
        bool inserted = false;
        for (auto iter = priority.begin(); iter != priority.end(); iter++) {
            // 对于非队首和队尾的特殊情况
            if (iter != priority.end() - 1 && iter != priority.begin()) {
                if (iter->prob < pt.prob && (iter - 1)->prob >= pt.prob) {
                    priority.insert(iter, pt);
                    inserted = true;
                    break;
                }
            }
            if (iter == priority.end() - 1) {
                if (iter->prob < pt.prob) {
                    priority.insert(iter, pt);
                } else {
                    priority.emplace_back(pt);
                }
                inserted = true;
                break;
            }
            if (iter == priority.begin() && iter->prob < pt.prob) {
                priority.insert(iter, pt);
                inserted = true;
                break;
            }
        }
        if (!inserted) {
            priority.emplace_back(pt);
        }
    }

    // 现在队首的PT善后工作已经结束，将其出队（删除）
    priority.erase(priority.begin());
}